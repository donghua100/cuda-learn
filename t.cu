#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define DATA_SIZE 1024*1024
#define THRREADS_NUM 256
#define BLOCK_NUM 32

int data[DATA_SIZE];

void generate_nums(int *nums, int size) {
	for (int i = 0; i < size; i++) {
		nums[i] = rand() %	10;
	}
}


int init_cuda() {
	int count;
	hipGetDeviceCount(&count);
	if (count == 0) {
		fprintf(stderr, "There is no device\n");
		return -1;
	}
	printf("There are %d device.\n", count);
	int i;
	for (i = 0; i < count; i++) {
		struct hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) break;
		}
	}
	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return -1;
	}
	hipSetDevice(i);
	return 0;
}

__global__ static void sumOfSquares(int *nums, int *result) {
	extern __shared__ int shared [];

	const int tid = threadIdx.x;
	const int bid = blockIdx.x;

	shared[tid] = 0;

	for (int i = bid * THRREADS_NUM + tid; i < DATA_SIZE; i += BLOCK_NUM * THRREADS_NUM) {
		shared[tid] += nums[i] * nums[i];
	}
	__syncthreads();
	if (tid == 0) {
		for (int i = 1; i < THRREADS_NUM; i++) {
			shared[0] += shared[i];
		}
	    result[bid] = shared[0];
	}
}

int main() {
	if (init_cuda() == 0) {
		printf("CUDA initialized.\n");
	}
	else {
		printf("initialized CUDA fail!\n");
		return -1;
	}
	generate_nums(data, DATA_SIZE);

	int *gpudata, *result;
	hipMalloc((void **)&gpudata, sizeof(int)*DATA_SIZE);
	hipMalloc((void **)&result, sizeof(int)*BLOCK_NUM);
	hipMemcpy(gpudata, data, sizeof(int)*DATA_SIZE, hipMemcpyHostToDevice);

	clock_t start = clock();
	sumOfSquares<<<BLOCK_NUM,THRREADS_NUM,sizeof(int) * THRREADS_NUM>>>(gpudata, result);
	hipDeviceSynchronize();
	clock_t end = clock();

	int sum[BLOCK_NUM];
	hipMemcpy(&sum, result, sizeof(int) * BLOCK_NUM, hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);


	int final_sum = 0;
	for (int i = 0; i < BLOCK_NUM; i++) final_sum += sum[i];

	printf("(GPU) sum = %d, using time: %lf ms\n", final_sum, (double)(end - start)/CLOCKS_PER_SEC*1000);

	final_sum = 0;
	start = clock();
	for (int i = 0; i < DATA_SIZE; i++) {
		final_sum += data[i] * data[i];
	}
	end = clock();
	// printf("(CPU) sum = %d, using time: %ld\n", sum, time_used);
	printf("(CPU) sum = %d, using time: %lf ms\n", final_sum, (double)(end - start)/CLOCKS_PER_SEC*1000);

	return 0;
}
